
#include <hip/hip_runtime.h>
extern "C" __global__ void timeReduction(const float *input, float *output, clock_t *timer)
{
  extern __shared__ float shared[];

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  if (tid == 0)
  {
    timer[bid] = clock();
  }

  // copy input
  shared[tid] = input[tid];
  shared[tid + blockDim.x] = input[tid + blockDim.x];

  // preform reduction to find minimum
  for (int d = blockDim.x; d > 0; d /= 2)
  {
    __syncthreads();

    if (tid < d)
    {
      float f0 = shared[tid];
      float f1 = shared[tid + d];

      if (f1 < f0)
      {
        shared[tid] = f1;
      }
    }
  }

  // write result
  if (tid == 0)
  {
    output[bid] = shared[0];
  }

  __syncthreads();

  if (tid == 0)
  {
    timer[bid + gridDim.x] = clock();
  }
}
